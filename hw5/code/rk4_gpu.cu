#include <stdio.h>
#include <time.h>
#include <sys/time.h> 
#include <hip/hip_runtime.h>

// System size
#define		N	4096

// Structures for profilling
struct timeval startTime;
struct timeval finishTime;
double timeIntervalLength;

// First RK step
__global__ void CUDA_rk4_0(const double h, const double pow[], const double c[], const double y[], double k[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (my_tid < N){
		int j;
		double my_k = pow[my_tid];
		for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*y[j];
		my_k *= h;
		k[my_tid] = my_k;
	}
}

// Second and third RK steps
__global__ void CUDA_rk4_1(const double h, const double pow[], const double c[], const double y[], const double k_old[], double k_new[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (my_tid < N){
		int j;
		double my_k = pow[my_tid];
	        for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*(y[j]+0.5*k_old[j]);
		my_k *= h;
		k_new[my_tid] = my_k;
	}
}

// Fourth RK step
__global__ void CUDA_rk4_2(const double h, const double pow[], const double c[], const double y[], 
						   const double k1[], const double k2[], const double k3[], double k4[], double yout[]){
	int my_tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (my_tid < N){
		int j;
		double my_k = pow[my_tid];
	        for (j = 0; j < N; j++)
			my_k -= c[my_tid*N+j]*(y[j]+k3[j]);
		my_k *= h;
		k4[my_tid] = my_k;

		yout[my_tid] = y[my_tid] + (k1[my_tid] + 2*k2[my_tid] + 2*k3[my_tid] + k4[my_tid])/6.0;
	}
}

int main(int argc, char* argv[]){

	// Define variables
	int i, j;
	double h, totalSum;
	double*  y;
	double*  k1;
	double*  k2;
	double*  k3;
	double*  k4;
	double*  pow;
	double*  yout;
	double*  c;

	// Allocate arrays
	hipMallocManaged(&y, N*sizeof(double));
	hipMallocManaged(&k1, N*sizeof(double));
	hipMallocManaged(&k2, N*sizeof(double));
	hipMallocManaged(&k3, N*sizeof(double));
	hipMallocManaged(&k4, N*sizeof(double));
	hipMallocManaged(&pow, N*sizeof(double));
	hipMallocManaged(&yout, N*sizeof(double));
	hipMallocManaged(&c, N*N*sizeof(double));

	// Initialize variables
	h = 0.3154;
	totalSum = 0.0;
	for (i = 0; i < N; i++){
		y[i] = i*i;
		pow[i] = 2*i;
		for (j = 0; j < N; j++)
			c[i*N+j] = i*i+j;
	}
	
	// Get the start time
	gettimeofday(&startTime, NULL);

	// Run each RK step in a separate kernel and synchronize device in between
	CUDA_rk4_0<<<N/1024,1024>>>(h, pow, c, y, k1);
	hipDeviceSynchronize();	

	CUDA_rk4_1<<<N/1024,1024>>>(h, pow, c, y, k1, k2);
	hipDeviceSynchronize();	

	CUDA_rk4_1<<<N/1024,1024>>>(h, pow, c, y, k2, k3);
	hipDeviceSynchronize();	

	CUDA_rk4_2<<<N/1024,1024>>>(h, pow, c, y, k1, k2, k3, k4, yout);
	hipDeviceSynchronize();

	// Get the end time
	gettimeofday(&finishTime, NULL);

	// Check results
	if (argc < 2){
		for (i = 0; i < N; i++)
			totalSum += yout[i];
		printf("Total Sum : %g \n", totalSum);
	}
	
	// Calculate the interval length 
	timeIntervalLength = (double)(finishTime.tv_sec-startTime.tv_sec)*1000000 
	                   + (double)(finishTime.tv_usec-startTime.tv_usec);
	timeIntervalLength = timeIntervalLength/1000;

	// Print the interval length
	if (argc < 2){
		printf("Interval length: %g msec.\n", timeIntervalLength);
	} else { 
		printf("%g\n", timeIntervalLength);
	}

	// Free memory
	hipFree(y);
	hipFree(k1);
	hipFree(k2);
	hipFree(k3);
	hipFree(k4);
	hipFree(pow);
	hipFree(yout);
	hipFree(c);

	return 0;
}
